#include "hip/hip_runtime.h"
#include "CrossEntropyWithSM.h"
#include "../../../commonGPU.cuh"

namespace Aoba
{
	namespace
	{
		__global__ void calcLoss(f32* forwardResult, f32* correctData, f32* dInput, f32* loss, u32 batchSize, u32 width, u32 dataSize)
		{
			u32 batchID = blockIdx.x * blockDim.x + threadIdx.x;

			if (batchID >= batchSize)
			{
				return;
			}

			u32 offset = batchID * dataSize;
			u32 correct = static_cast<u32>(correctData[batchID]);

			f32 max = forwardResult[offset + 0];
			u32 maxIndex = 0;
			f32 sum = 0.0f;
			for (u32 i = 0; i < width; i++)
			{
				f32 cand = forwardResult[offset + i];
				if (max < cand)
				{
					max = cand;
					maxIndex = i;
				}
			}

			for (u32 i = 0; i < width; i++)
			{
				sum += exp(forwardResult[offset + i] - max);
			}

			for (u32 i = 0; i < width; i++)
			{
				dInput[offset + i] = ((exp(forwardResult[offset + i] - max) / sum) - (correct == i ? 1 : 0)) / batchSize;
			}
			
			loss[batchID] = -log(exp(forwardResult[offset + correct] - max) / sum + 1e-7);
		}
	}

	namespace lossFunction
	{
		void CrossEntropyWithSM::mallocOnGPU()
		{
			mDInputDataOnGPU.size =mBatchSize * mTrainingDataShape.width;
			CHECK(hipMalloc((void**)(&(mDInputDataOnGPU.address)), mDInputDataOnGPU.size * sizeof(f32)));

			mLossTblOnGPU.size = mBatchSize;
			CHECK(hipMalloc((void**)(&mLossTblOnGPU.address), mLossTblOnGPU.size * sizeof(f32)));
		}

		f32 CrossEntropyWithSM::calcLossAndDInputOnGPU()
		{
			dim3 block(16, 1);
			dim3 grid(
				(mBatchSize + block.x - 1) / block.x);

			//�G���[���o��i����̂��߂Ɏc���Ă����j
			/*f32* lossTblOnGPU = nullptr;
			CHECK(hipMalloc((void**)(&lossTblOnGPU), mDataShape.batchSize * sizeof(f32)));*/
#if _DEBUG
			assert(mBatchSize != 0);
#endif
			calcLoss<<<grid, block>>>(
				mForwardResultOnGPU->address,
				mCorrectDataOnGPU->address,
				mDInputDataOnGPU.address, 
				mLossTblOnGPU.address, 
				mBatchSize,
				mTrainingDataShape.width,
				mForwardResultOnGPU->size / mBatchSize);
#if _DEBUG
			CHECK(hipDeviceSynchronize());
#endif
			f32 loss = 0;
			std::vector<f32> lossOnCPU(mLossTblOnGPU.size);
			CHECK(hipMemcpy(lossOnCPU.data(), mLossTblOnGPU.address, mLossTblOnGPU.size * sizeof(f32), hipMemcpyDeviceToHost));

			for (u32 i = 0; i < mLossTblOnGPU.size; i++)
			{
				loss += lossOnCPU[i];
			}


			return loss / mBatchSize;
		}
	}
}