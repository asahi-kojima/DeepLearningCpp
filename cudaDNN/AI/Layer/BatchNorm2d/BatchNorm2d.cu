#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>
#include <cassert>

#include "BatchNorm2d.h"
#include "../../../commonOnlyGPU.cuh"

namespace Aoba {
	namespace layer
	{
		namespace
		{
			//__global__ void BatchNorm2d_forwardOnGPU(
			//	f32* input, 
			//	f32* intermediateResult,
			//	f32* forwardResult, 
			//	f32* Gamma, 
			//	f32* Beta,
			//	f32* Sigma,
			//	u32 batchSize,
			//	u32 channel,
			//	u32 height,
			//	u32 width)
			//{
			//	u32 c = blockIdx.x * blockDim.x + threadIdx.x;
			//	
			//	if (c >= channel)
			//	{
			//		return;
			//	}



			//	f32 ep = 1e-7;
			//	u32 hXw = height * width;
			//	u32 cXhXw = channel * hXw;
			//	f32 mean = 0.0f;
			//	f32 sigma = 0.0f;

			//	//------------------------------------------------------------------
			//	//���ς��v�Z
			//	//------------------------------------------------------------------
			//	for (u32 N = 0; N < batchSize; N++)
			//	{
			//		for (u32 hw = 0; hw < hXw; hw++)
			//		{
			//			mean += input[N * cXhXw + c * hXw + hw];
			//		}
			//	}
			//	mean /= (batchSize * hXw);

			//	//------------------------------------------------------------------
			//	//�΍����v�Z
			//	//------------------------------------------------------------------
			//	f32 sigma2 = 0.0f;
			//	for (u32 N = 0; N < batchSize; N++)
			//	{
			//		for (u32 hw = 0; hw < hXw; hw++)
			//		{
			//			f32 diff = input[N * cXhXw + c * hXw + hw] - mean;
			//			sigma2 += diff * diff;
			//		}
			//}
			//	sigma2 /= (batchSize * hXw);
			//	sigma = std::sqrt(sigma2);

			//	//------------------------------------------------------------------
			//	//�W����
			//	//------------------------------------------------------------------
			//	f32 gamma = Gamma[c];
			//	f32 beta = Beta[c];
			//	for (u32 N = 0; N < batchSize; N++)
			//	{
			//		for (u32 hw = 0; hw < hXw; hw++)
			//		{
			//			u32 index = N * cXhXw + c * hXw + hw;
			//			f32 normalizeResult = (input[index] - mean) / sigma;
			//			intermediateResult[index] = normalizeResult;
			//			forwardResult[index] = gamma * normalizeResult + beta;
			//		}
			//	}

			//	Sigma[c] = sigma;
			//}

			__global__ void BatchNorm2d_forwardOnGPU(
				f32* input,
				f32* intermediateResult,
				f32* forwardResult,
				f32* Gamma,
				f32* Beta,
				f32* Sigma,
				u32 batchSize,
				u32 channel,
				u32 height,
				u32 width)
			{
				u32 c = blockIdx.x * blockDim.x + threadIdx.x;

				if (c >= channel)
				{
					return;
				}



				f32 ep = 1e-7;
				u32 hXw = height * width;
				u32 cXhXw = channel * hXw;
				f32 mean = 0.0f;
				f32 sigma = 0.0f;

				//------------------------------------------------------------------
				//���ς��v�Z
				//------------------------------------------------------------------
				for (u32 N = 0; N < batchSize; N++)
				{
					for (u32 hw = 0; hw < hXw; hw++)
					{
						mean += input[N * cXhXw + c * hXw + hw];
					}
				}
				mean /= (batchSize * hXw);

				//------------------------------------------------------------------
				//�΍����v�Z
				//------------------------------------------------------------------
				f32 sigma2 = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
					for (u32 hw = 0; hw < hXw; hw++)
					{
						f32 diff = input[N * cXhXw + c * hXw + hw] - mean;
						sigma2 += diff * diff;
					}
				}
				sigma2 /= (batchSize * hXw);
				sigma = std::sqrt(sigma2);

				//------------------------------------------------------------------
				//�W����
				//------------------------------------------------------------------
				f32 gamma = Gamma[c];
				f32 beta = Beta[c];
				for (u32 N = 0; N < batchSize; N++)
				{
					for (u32 hw = 0; hw < hXw; hw++)
					{
						u32 index = N * cXhXw + c * hXw + hw;
						f32 normalizeResult = (input[index] - mean) / sigma;
						intermediateResult[index] = normalizeResult;
						forwardResult[index] = gamma * normalizeResult + beta;
					}
				}

				Sigma[c] = sigma;
			}

			__global__ void BatchNorm2d_backwardOnGPU(
				f32* dout,
				f32* intermediateResult,
				f32* backwardResult,
				f32* Gamma,
				f32* DGamma,
				f32* DBeta,
				f32* Sigma,
				u32 batchSize,
				u32 channel,
				u32 height,
				u32 width)
			{
				u32 c = blockIdx.x * blockDim.x + threadIdx.x;

				if (c >= channel)
				{
					return;
				}



				f32 ep = 1e-7;
				u32 hXw = height * width;
				u32 cXhXw = channel * hXw;

				f32 dGamma = 0.0f;
				f32 dBeta = 0.0f;

				for (u32 N = 0; N < batchSize; N++)
				{
					for (u32 hw = 0; hw < hXw; hw++)
					{
						u32 index = N * cXhXw + c * hXw + hw;
						f32 dO = dout[index];
						dGamma += dO * intermediateResult[index];
						dBeta += dO;
					}
				}
				DGamma[c] = dGamma;
				DBeta[c] = dBeta;



				f32 dMean = 0.0f;
				f32 diMean = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
					for (u32 hw = 0; hw < hXw; hw++)
					{
						u32 index = N * cXhXw + c * hXw + hw;
						dMean += dout[index];
						diMean += dout[index] * intermediateResult[index];
					}
				}
				dMean /= (batchSize * hXw);
				diMean /= (batchSize * hXw);

				for (u32 N = 0; N < batchSize; N++)
				{
					for (u32 hw = 0; hw < hXw; hw++)
					{
						u32 index = N * cXhXw + c * hXw + hw;
						backwardResult[index] = (Gamma[c] / (Sigma[c] + 1e-7)) * (dout[index] - dMean - intermediateResult[index] * diMean);
					}
				}
			}
		}


		void BatchNorm2d::mallocOnGPU()
		{
			mParametersPtrOnGPU.resize(2);
			mDParametersPtrOnGPU.resize(2);


			//------------------------------------------------------------------
			//Gamma
			//------------------------------------------------------------------
			DataArray& gammaParam = mParametersPtrOnGPU[0];
			DataArray& gammaDParam = mDParametersPtrOnGPU[0];

			gammaParam.size = mDataShape.channel;
			gammaDParam.size = gammaParam.size;

			CHECK(hipMalloc((void**)(&(gammaParam.address)), gammaParam.size * sizeof(f32)));
			CHECK(hipMalloc((void**)(&(gammaDParam.address)), gammaDParam.size * sizeof(f32)));
			{
				f32* tmp = new f32[gammaParam.size];
				for (u32 i = 0; i < gammaParam.size; i++)
				{
					tmp[i] = 1.0f;
				}
				CHECK(hipMemcpy(gammaParam.address, tmp, gammaParam.size * sizeof(f32), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(gammaDParam.address, tmp, gammaDParam.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			 
			//------------------------------------------------------------------
			//Beta
			//------------------------------------------------------------------
			DataArray& betaParam = mParametersPtrOnGPU[1];
			DataArray& betaDParam = mDParametersPtrOnGPU[1];

			betaParam.size = mDataShape.channel;
			betaDParam.size = betaParam.size;

			CHECK(hipMalloc((void**)(&(betaParam.address)), betaParam.size * sizeof(f32)));
			CHECK(hipMalloc((void**)(&(betaDParam.address)), betaDParam.size * sizeof(f32)));

			{
				f32* tmp = new f32[betaParam.size];
				for (u32 i = 0; i < betaParam.size; i++)
				{
					tmp[i] = 0.0f;
				}
				CHECK(hipMemcpy(betaParam.address, tmp, betaParam.size * sizeof(f32), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(betaDParam.address, tmp, betaDParam.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			//------------------------------------------------------------------
			//Sigma
			//------------------------------------------------------------------
			mSigmaOnGPU.size = mDataShape.channel;
			mSigmaOnGPU.byteSize = mSigmaOnGPU.size * sizeof(f32);

			CHECK(hipMalloc((void**)(&(mSigmaOnGPU.address)), mSigmaOnGPU.size * sizeof(f32)));

			{
				f32* tmp = new f32[mSigmaOnGPU.size];
				for (u32 i = 0; i < mSigmaOnGPU.size; i++)
				{
					tmp[i] = 0.0f;
				}
				CHECK(hipMemcpy(mSigmaOnGPU.address, tmp, mSigmaOnGPU.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			//------------------------------------------------------------------
			//�`���p
			//------------------------------------------------------------------
			mForwardResultOnGPU.size = mBatchSize * mDataShape.getDataSize();
			mBackwardResultOnGPU.size = mForwardResultOnGPU.size;
			mIntermediateResultOnGPU.size = mForwardResultOnGPU.size;

			mForwardResultOnGPU.address = new f32[mForwardResultOnGPU.size];
			mBackwardResultOnGPU.address = new f32[mBackwardResultOnGPU.size];
			mIntermediateResultOnGPU.address = new f32[mIntermediateResultOnGPU.size];
			CHECK(hipMalloc((void**)(&(mForwardResultOnGPU.address)), mForwardResultOnGPU.size * sizeof(f32)));
			CHECK(hipMalloc((void**)(&(mBackwardResultOnGPU.address)), mBackwardResultOnGPU.size * sizeof(f32)));
			CHECK(hipMalloc((void**)(&(mIntermediateResultOnGPU.address)), mIntermediateResultOnGPU.size * sizeof(f32)));

			{
				f32* tmp = new f32[mForwardResultOnGPU.size];
				for (u32 i = 0; i < mForwardResultOnGPU.size; i++)
				{
					tmp[i] = 0.0f;
				}
				CHECK(hipMemcpy(mForwardResultOnGPU.address, tmp, mForwardResultOnGPU.size * sizeof(f32), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(mBackwardResultOnGPU.address, tmp, mBackwardResultOnGPU.size * sizeof(f32), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(mIntermediateResultOnGPU.address, tmp, mIntermediateResultOnGPU.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;
			}
		}

		void BatchNorm2d::forwardOnGPU()
		{
			dim3 block(16);
			dim3 grid((mDataShape.channel + block.x - 1) / block.x);

			BatchNorm2d_forwardOnGPU << <grid, block >> > (
				mInputDataOnGPU->address,
				mIntermediateResultOnGPU.address,
				mForwardResultOnGPU.address,
				mParametersPtrOnGPU[0].address,
				mParametersPtrOnGPU[1].address,
				mSigmaOnGPU.address,
				mBatchSize,
				mDataShape.channel,
				mDataShape.height,
				mDataShape.width);

#if _DEBUG
			CHECK(hipDeviceSynchronize());
#endif
		}

		void BatchNorm2d::backwardOnGPU()
		{
			dim3 block(16);
			dim3 grid((mDataShape.channel + block.x - 1) / block.x);

			BatchNorm2d_backwardOnGPU << <grid, block >> > (
				mDInputDataOnGPU->address,
				mIntermediateResultOnGPU.address,
				mBackwardResultOnGPU.address,
				mParametersPtrOnGPU[0].address,
				mDParametersPtrOnGPU[0].address,
				mDParametersPtrOnGPU[1].address,
				mSigmaOnGPU.address,
				mBatchSize,
				mDataShape.channel,
				mDataShape.height,
				mDataShape.width);

#if _DEBUG
			CHECK(hipDeviceSynchronize());
#endif
		}

		void BatchNorm2d::terminateOnGPU()
		{

		}
	}
}