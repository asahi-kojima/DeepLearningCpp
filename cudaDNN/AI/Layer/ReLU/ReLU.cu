#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>
#include <cassert>

#include "ReLU.h"

namespace Aoba {
	namespace layer
	{
		namespace
		{
			__global__ void ReLUForward(
				f32* y, f32* x,
				f32* mask, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= outputSize || yid >= batchSize)
				{
					return;
				}

				u32 id = yid * outputSize + xid;
#if INDEX_DEBUG
				if (id >= outputSize * batchSize)
				{
					printf("ReLU  : out of range : %d", id);
					assert(0);
				}
#endif
				f32 input = x[id];
				if (input < 0)
				{
					mask[id] = 0;
					y[id] = 0;
				}
				else
				{
					mask[id] = 1;
					y[id] = input;
				}
				//printf("%lf\n",y[id]);
			}



			__global__ void ReLUBackward(f32* y, f32* x,
				f32* mask, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= inputSize || yid >= batchSize)
				{
					return;
				}

				u32 id = yid * inputSize + xid;
#if INDEX_DEBUG
				if (id >= inputSize * batchSize)
				{
					printf("ReLU  : out of range : %d", id);
					assert(0);
				}
#endif
				y[id] = x[id] * mask[id];
			}
		}


		void ReLU::mallocOnGPU()
		{
			mMaskOnGPU.size = mBatchSize * mDataSize;
			MALLOC_ON_GPU(mMaskOnGPU);
			INITIALIZE_GPU_DATA_1(mMaskOnGPU);

			//�v�Z���ʂ��i�[���邽�߂̃������m��
			mForwardResultOnGPU.size = mBackwardResultOnGPU.size = mBatchSize * mDataSize;
			MALLOC_ON_GPU(mForwardResultOnGPU);
			MALLOC_ON_GPU(mBackwardResultOnGPU);
			INITIALIZE_GPU_DATA_0(mForwardResultOnGPU);
			INITIALIZE_GPU_DATA_0(mBackwardResultOnGPU);
		}

		void ReLU::forwardOnGPU()
		{
			dim3 block(16, 16);
			dim3 grid(
				(mDataSize + block.x - 1) / block.x,
				(mBatchSize + block.y - 1) / block.y);
#if TIME_DEBUG
			{
				std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif
				ReLUForward << <grid, block >> > (
					mForwardResultOnGPU.address,
					mInputDataOnGPU->address,
					mMaskOnGPU.address,
					mDataSize,
					mDataSize,
					mBatchSize);
#if GPU_SYNC_DEBUG
				CHECK(hipDeviceSynchronize());
#endif
#if TIME_DEBUG
				f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
				std::string name = "";
				(((name += __FUNCTION__) += " : ") += std::to_string(mInstanceID)) += " : ReLUForward";
				timers[name] = elapsedTime;
			}
#endif
		}

		void ReLU::backwardOnGPU()
		{
			dim3 block(16, 16);
			dim3 grid(
				(mDataSize + block.x - 1) / block.x,
				(mBatchSize + block.y - 1) / block.y);
#if TIME_DEBUG
			{
				std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif
				ReLUBackward << <grid, block >> > (
					mBackwardResultOnGPU.address,
					mDInputDataOnGPU->address,
					mMaskOnGPU.address,
					mDataSize,
					mDataSize,
					mBatchSize);
#if GPU_SYNC_DEBUG
				CHECK(hipDeviceSynchronize());
#endif
#if TIME_DEBUG
				f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
				std::string name = "";
				(((name += __FUNCTION__) += " : ") += std::to_string(mInstanceID)) += " : ReLUBackward";
				timers[name] = elapsedTime;
			}
#endif
		}

		void ReLU::terminateOnGPU()
		{

		}
	}
}