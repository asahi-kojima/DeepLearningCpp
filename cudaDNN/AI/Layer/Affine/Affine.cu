#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>
#include <cassert>

#include "Affine.h"
#include "../../../commonGPU.cuh"

namespace Aoba {
	namespace layer
	{
		namespace
		{


			__global__ void AffineForward(
				f32* y, f32* A,
				f32* x, f32* b, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= outputSize || yid >= batchSize)
				{
					return;
				}
				u32 id = yid * outputSize + xid;

				f32 result = 0.0f;
				for (u32 i = 0; i < inputSize; i++)
				{
#if _DEBUG
					u32 tmp = xid * inputSize + i;
					if (tmp < 0 || tmp >= inputSize * outputSize)
					{
						printf("Affine A parameter : out of range : %d\n", tmp);
						printf("threadId x = %d  ,  y = %d\n", threadIdx.x, threadIdx.y);
						assert(0);
					}
					tmp = yid * inputSize + i;
					if (tmp < 0 || tmp >= inputSize * batchSize)
					{
						printf("Affine x parameter : out of range : %d", tmp);
						assert(0);
					}
#endif
					result += A[xid * inputSize + i] * x[yid * inputSize + i];
					if (xid == 0)
						printf("%lf\n", x[yid * inputSize + i]);
				}
#if _DEBUG
				if (!(id >= 0 && id < batchSize * outputSize))
				{
					printf("Affine y parameter : out of range : %d", id);
					assert(0);
				}
#endif
				y[id] = result + b[xid];
			}

			__global__ void AffineBackward(f32* dA, f32* dout, f32* input, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= inputSize || yid >= outputSize)
				{
					return;
				}

				u32 id = yid * inputSize + xid;

				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if (N * inputSize + xid >= batchSize * inputSize)
					{
						assert(0);
					}
					if (N * outputSize + yid >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += dout[N * outputSize + yid] * input[N * inputSize + xid];
				}

				dA[id] = result;
				//printf("dA[%d]=%lf\n", id,result);
			}

			__global__ void biasBackward(f32* dBias, f32* dout, u32 outputSize, u32 batchSize)
			{
				u32 id = blockIdx.x * blockDim.x + threadIdx.x;
				if (id >= outputSize)
				{
					return;
				}
				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if ((N * outputSize + id) >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += dout[N * outputSize + id];
				}
#if _DEBUG
				if (id >= outputSize)
				{
					assert(0);
				}
#endif
				dBias[id] = result;
				//printf("%lf\n", result);
			}

			__global__ void doutBackward(f32* dOut, f32* A, f32* dIn, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;//input
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;//batch

				if (xid >= inputSize || yid >= batchSize)
				{
					return;
				}

				f32 result = 0.0f;
				for (u32 i = 0; i < outputSize; i++)
				{
#if _DEBUG
					if (i * inputSize + xid >= outputSize * inputSize)
					{
						assert(0);
					}
					if (yid * outputSize + i >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += A[i * inputSize + xid] * dIn[yid * outputSize + i];
				}
				dOut[yid * inputSize + xid] = result;
				//printf("dOut[%d * %d + %d] = %lf\n",yid, inputSize, xid, dOut[yid * inputSize + xid]);
			}
		}
		void Affine::mallocOnGPU()
		{
			pParametersOnGPU.resize(2);
			pDParametersOnGPU.resize(2);

			//Affine�p�����[�^
			paramMemory& affineParam = pParametersOnGPU[0];
			paramMemory& affineDParam = pDParametersOnGPU[0];

			affineParam.size = affineDParam.size = mOutputSize * mInputSize;

			CHECK(hipMalloc((void**)(&(affineParam.address)), affineParam.size * sizeof(f32)));
			CHECK(hipMalloc((void**)(&(affineDParam.address)), affineDParam.size * sizeof(f32)));

			f32* tmpAffineParam = new f32[affineParam.size];
			{
				std::random_device seed_gen;
				std::default_random_engine engine(seed_gen());
				std::normal_distribution<> dist(0.0, std::sqrt(2.0 / mInputSize));

				std::vector<f32> tmp(affineParam.size);
				for (u32 idx = 0; idx < affineParam.size; idx++)
				{
					tmp[idx] = mAffineParamWeight * static_cast<f32>(dist(engine));
				}
				CHECK(hipMemcpy(affineParam.address, tmp.data(), affineParam.size * sizeof(f32), hipMemcpyHostToDevice));

				for (u32 idx = 0; idx < affineDParam.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(affineDParam.address, tmp.data(), affineDParam.size * sizeof(f32), hipMemcpyHostToDevice));
			}


			//Bias�p�����[�^
			paramMemory& biasParam = pParametersOnGPU[1];
			paramMemory& biasDParam = pDParametersOnGPU[1];

			biasParam.size = biasDParam.size = mOutputSize;

			hipMalloc((void**)(&(biasParam.address)), biasParam.size * sizeof(f32));
			hipMalloc((void**)(&(biasDParam.address)), biasDParam.size * sizeof(f32));
			{
				f32* tmp = new f32[biasParam.size];
				for (u32 idx = 0; idx < biasParam.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(biasParam.address, tmp, biasParam.size * sizeof(f32), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(biasDParam.address, tmp, biasDParam.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			//�v�Z���ʂ��i�[���邽�߂̃������m��
			mForwardResultOnGPU.size = mBatchSize * mOutputSize;
			mBackwardResultOnGPU.size = mBatchSize * mInputSize;
			CHECK(hipMalloc((void**)(&(mForwardResultOnGPU.address)),
				mForwardResultOnGPU.size * sizeof(f32)));
			CHECK(hipMalloc((void**)(&(mBackwardResultOnGPU.address)),
				mBackwardResultOnGPU.size * sizeof(f32)));
			{
				f32* tmp = new f32[mForwardResultOnGPU.size];
				for (u32 idx = 0; idx < mForwardResultOnGPU.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mForwardResultOnGPU.address, tmp,
					mForwardResultOnGPU.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;


				tmp = new f32[mBackwardResultOnGPU.size];
				for (u32 idx = 0; idx < mBackwardResultOnGPU.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mBackwardResultOnGPU.address, tmp,
					mBackwardResultOnGPU.size * sizeof(f32), hipMemcpyHostToDevice));
				delete[] tmp;
			}
		}

		void Affine::forwardOnGPU()
		{
			dim3 block(16, 16);
			dim3 grid(
				(mOutputSize + block.x - 1) / block.x,
				(mBatchSize + block.y - 1) / block.y);

			AffineForward << <grid, block >> > (
				mForwardResultOnGPU.address,
				pParametersOnGPU[0].address,
				mInputDataOnGPU->address,
				pParametersOnGPU[1].address,
				mOutputSize,
				mInputSize,
				mBatchSize);
		}

		void Affine::backwardOnGPU()
		{
			//dout�̋t�`��
			{
				dim3 block(16, 16);
				dim3 grid(
					(mInputSize + block.x - 1) / block.x,
					(mBatchSize + block.y - 1) / block.y);
				doutBackward << <grid, block >> > (
					mBackwardResultOnGPU.address,
					pParametersOnGPU[0].address,
					mDInputDataOnGPU->address,
					mOutputSize,
					mInputSize,
					mBatchSize);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}

			//A�̋t�`��
			{
				dim3 block(16, 16);
				dim3 grid(
					(mInputSize + block.x - 1) / block.x,
					(mOutputSize + block.y - 1) / block.y);

				AffineBackward << <grid, block >> > (
					pDParametersOnGPU[0].address,
					mDInputDataOnGPU->address,
					mInputDataOnGPU->address,
					mOutputSize,
					mInputSize,
					mBatchSize);
			}

			//Bias�̋t�`��
			{
				dim3 block(16);
				dim3 grid((mOutputSize + block.x - 1) / block.x);

				biasBackward << <grid, block >> > (
					pDParametersOnGPU[1].address,
					mDInputDataOnGPU->address,
					mOutputSize,
					mBatchSize);
			}
		}

		void Affine::terminateOnGPU()
		{

		}

	}
}