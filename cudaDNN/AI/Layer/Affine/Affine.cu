#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>
#include <cassert>

#include "Affine.h"
#include "../../../commonGPU.cuh"

namespace Aoba {
	namespace layer
	{
		namespace
		{
			using flowDataType = BaseLayer::flowDataType;


			__global__ void AffineForward(
				flowDataType* y, flowDataType* A, 
				flowDataType* x, flowDataType* b, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= outputSize || yid >= batchSize)
				{
					return;
				}
				u32 id = yid * outputSize + xid;

				f32 result = 0.0f;
				for (u32 i = 0; i < inputSize; i++)
				{
#if _DEBUG
					u32 tmp = xid * inputSize + i;
					if (tmp < 0 || tmp >= inputSize * outputSize)
					{
						printf("Affine A parameter : out of range : %d\n", tmp);
						printf("threadId x = %d  ,  y = %d\n", threadIdx.x, threadIdx.y);
						assert(0);
					}
					tmp = yid * inputSize + i;
					if (tmp < 0 || tmp >= inputSize * batchSize)
					{
						printf("Affine x parameter : out of range : %d", tmp);
						assert(0);
					}
#endif
					result += A[xid * inputSize + i] * x[yid * inputSize + i]; 
				}
#if _DEBUG
				if (!(id >= 0 && id < batchSize * outputSize))
				{
					printf("Affine y parameter : out of range : %d", id);
					assert(0);
				}
#endif
				y[id] = result + b[xid];
			}

			__global__ void AffineBackward(flowDataType* dA, flowDataType* dout, flowDataType* input, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= inputSize || yid >= outputSize)
				{
					return;
				}

				u32 id = yid * inputSize + xid;

				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if (N * inputSize + xid >= batchSize * inputSize)
					{
						assert(0);
					}
					if (N * outputSize + yid >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += input[N * inputSize + xid] * dout[N * outputSize + yid];
				}

				dA[id] = result;
			}

			__global__ void biasBackward(flowDataType * dBias, flowDataType * dout, u32 outputSize,u32 batchSize)
			{
				u32 id = blockIdx.x * blockDim.x + threadIdx.x;
				if (id >= outputSize)
				{
					return;
				}
				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if ((N * outputSize + id) >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += dout[N * outputSize + id];
				}
#if _DEBUG
				if (id >= outputSize)
				{
					assert(0);
				}
#endif
				dBias[id] = result;
			}

			__global__ void doutBackward(flowDataType* dOut ,flowDataType * A, flowDataType* dIn,u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;//input
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;//batch
				
				if (xid >= inputSize || yid >= batchSize)
				{
					return;
				}

				f32 result = 0.0f;
				for (u32 i = 0; i < outputSize; i++)
				{
#if _DEBUG
					if (i * inputSize + xid >= outputSize * inputSize)
					{
						assert(0);
					}
					if (yid * outputSize + i >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += A[i * inputSize + xid] * dIn[yid * outputSize + i];
				}
				dOut[yid * inputSize + xid] = result;
			}
		}
		void Affine::initializeOnGPU()
		{
			pParametersOnGPU.resize(2);
			pDParametersOnGPU.resize(2);

			//Affine�p�����[�^
			paramMemory& affineParam = pParametersOnGPU[0];
			paramMemory& affineDParam = pDParametersOnGPU[0];

			affineParam.size = affineDParam.size = mOutputSize * mInputSize;

			CHECK(hipMalloc((void**)(&(affineParam.address)), affineParam.size * sizeof(parameterType))   );
			CHECK(hipMalloc((void**)(&(affineDParam.address)), affineDParam.size * sizeof(parameterType)) );

			parameterType* tmpAffineParam = new parameterType[affineParam.size];
			{
				std::random_device seed_gen;
				std::default_random_engine engine(seed_gen());
				std::normal_distribution<> dist(0.0, std::sqrt(2.0 / mInputSize));

				parameterType* tmp = new parameterType[affineParam.size];
				for (u32 idx = 0; idx < affineParam.size; idx++)
				{
					tmp[idx] = mAffineParamWeight * static_cast<f32>(dist(engine)) / std::sqrt(2.0f / mInputSize);
				}
				CHECK(hipMemcpy(affineParam.address, tmp, affineParam.size * sizeof(parameterType), hipMemcpyHostToDevice));

				for (u32 idx = 0; idx < affineDParam.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(affineDParam.address, tmp, affineDParam.size * sizeof(parameterType), hipMemcpyHostToDevice));
				delete[] tmp;
			}


			//Bias�p�����[�^
			paramMemory& biasParam = pParametersOnGPU[1];
			paramMemory& biasDParam = pDParametersOnGPU[1];

			biasParam.size = biasDParam.size = mOutputSize;

			hipMalloc((void**)(&(biasParam.address)), biasParam.size * sizeof(parameterType));
			hipMalloc((void**)(&(biasDParam.address)), biasDParam.size * sizeof(parameterType));
			{
				parameterType* tmp = new parameterType[biasParam.size];
				for (u32 idx = 0; idx < biasParam.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(biasParam.address, tmp, biasParam.size * sizeof(parameterType), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(biasDParam.address, tmp, biasDParam.size * sizeof(parameterType), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			//�v�Z���ʂ��i�[���邽�߂̃������m��
			mForwardResultOnGPU.size = mBatchSize * mOutputSize;
			mBackwardResultOnGPU.size = mBatchSize * mInputSize;
			CHECK(hipMalloc((void**)(&(mForwardResultOnGPU.address)), 
				mForwardResultOnGPU.size * sizeof(flowDataType)));
			CHECK(hipMalloc((void**)(&(mBackwardResultOnGPU.address)), 
				mBackwardResultOnGPU.size * sizeof(flowDataType)));
			{
				flowDataType* tmp = new flowDataType[mForwardResultOnGPU.size];
				for (u32 idx = 0; idx < mForwardResultOnGPU.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mForwardResultOnGPU.address, tmp, 
					mForwardResultOnGPU.size * sizeof(flowDataType), hipMemcpyHostToDevice));
				delete[] tmp;


				tmp = new flowDataType[mBackwardResultOnGPU.size];
				for (u32 idx = 0; idx < mBackwardResultOnGPU.size; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mBackwardResultOnGPU.address, tmp, 
					mBackwardResultOnGPU.size * sizeof(flowDataType), hipMemcpyHostToDevice));
				delete[] tmp;
			}
		}

		void Affine::forwardOnGPU()
		{
			dim3 block(16,16);
			dim3 grid(
				(mOutputSize + block.x - 1) / block.x,
				(mBatchSize + block.y - 1) / block.y);

			AffineForward << <grid, block >> > (
				mForwardResultOnGPU.address,
				pParametersOnGPU[0].address,
				mInputDataOnGPU->address,
				pParametersOnGPU[1].address,
				mOutputSize,
				mInputSize,
				mBatchSize);
#if _DEBUG
			CHECK(hipDeviceSynchronize());
#endif
		}

		void Affine::backwardOnGPU()
		{
			//dout�̋t�`��
			{
				dim3 block(16, 16);
				dim3 grid(
					(mInputSize + block.x - 1) / block.x,
					(mBatchSize + block.y - 1) / block.y);
				doutBackward << <grid, block >> > (
					mBackwardResultOnGPU.address,
					pParametersOnGPU[0].address,
					mDInputDataOnGPU->address,
					mOutputSize,
					mInputSize,
					mBatchSize);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}

			//A�̋t�`��
			{
				dim3 block(16,16);
				dim3 grid(
					(mOutputSize + block.x - 1) / block.x,
					(mInputSize + block.y - 1) / block.y);
				AffineBackward << <grid, block >> > (
					pDParametersOnGPU[0].address,
					mDInputDataOnGPU->address,
					mInputDataOnGPU->address,
					mOutputSize,
					mInputSize,
					mBatchSize);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}

			//Bias�̋t�`��
			{
				dim3 block(16);
				dim3 grid((mOutputSize + block.x - 1) / block.x);
				biasBackward << <grid, block >> > (
					pDParametersOnGPU[1].address,
					mDInputDataOnGPU->address,
					mOutputSize,
					mBatchSize);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}
		}

		void Affine::terminateOnGPU()
		{

		}

	}
}