#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>
#include <cassert>

//���̃}�N����CUDA�t�@�C�����R���p�C������鎞�ɒ�`�����B
//�C���e���Z���X�̃G���[���ꎞ�I�ɗ}�����邽�߂ɂ��̒�`��u���Ă���B
#if !defined(__HIPCC__)
#define __HIPCC__
#endif

#include <hip/device_functions.h>


#include "Convolution.h"
#include "../../../commonOnlyGPU.cuh"
#include "../../../common.h"


namespace Aoba {
	namespace layer
	{
		namespace
		{


			__global__ void ConvolutionForward(
				f32* y, f32* A,
				f32* x, f32* b, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= outputSize || yid >= batchSize)
				{
					return;
				}
				u32 id = yid * outputSize + xid;

				f32 result = 0.0f;
				for (u32 i = 0; i < inputSize; i++)
				{
					//#if _DEBUG
					//					u32 tmp = xid * inputSize + i;
					//					if (tmp < 0 || tmp >= inputSize * outputSize)
					//					{
					//						printf("Affine A parameter : out of range : %d\n", tmp);
					//						printf("threadId x = %d  ,  y = %d\n", threadIdx.x, threadIdx.y);
					//						assert(0);
					//					}
					//					tmp = yid * inputSize + i;
					//					if (tmp < 0 || tmp >= inputSize * batchSize)
					//					{
					//						printf("Affine x parameter : out of range : %d", tmp);
					//						assert(0);
					//					}
					//#endif
					result += A[xid * inputSize + i] * x[yid * inputSize + i];
				}
				//#if _DEBUG
				//				if (!(id >= 0 && id < batchSize * outputSize))
				//				{
				//					printf("Affine y parameter : out of range : %d", id);
				//					assert(0);
				//				}
				//#endif
				y[id] = result + b[xid];
			}


			__global__ void FilterBackward(f32* dA, f32* dout, f32* input, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= inputSize || yid >= outputSize)
				{
					return;
				}

				u32 id = yid * inputSize + xid;

				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if (N * inputSize + xid >= batchSize * inputSize)
					{
						assert(0);
					}
					if (N * outputSize + yid >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += dout[N * outputSize + yid] * input[N * inputSize + xid];
				}

				dA[id] = result;
				//printf("dA[%d]=%lf\n", id,result);
			}

			__global__ void biasBackward(f32* dBias, f32* dout, u32 outputSize, u32 batchSize)
			{
				u32 id = blockIdx.x * blockDim.x + threadIdx.x;
				if (id >= outputSize)
				{
					return;
				}
				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if ((N * outputSize + id) >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += dout[N * outputSize + id];
				}
#if _DEBUG
				if (id >= outputSize)
				{
					assert(0);
				}
#endif
				dBias[id] = result;
				//printf("%lf\n", result);
			}

			__global__ void doutBackward(f32* dOut, f32* A, f32* dIn, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;//input
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;//batch

				if (xid >= inputSize || yid >= batchSize)
				{
					return;
				}

				f32 result = 0.0f;
				for (u32 i = 0; i < outputSize; i++)
				{
#if _DEBUG
					if (i * inputSize + xid >= outputSize * inputSize)
					{
						assert(0);
					}
					if (yid * outputSize + i >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += A[i * inputSize + xid] * dIn[yid * outputSize + i];
				}
				dOut[yid * inputSize + xid] = result;
				//printf("dOut[%d * %d + %d] = %lf\n",yid, inputSize, xid, dOut[yid * inputSize + xid]);
			}
		}
		void Convolution::mallocOnGPU()
		{
			mParametersPtrOnGPU.resize(2);
			mDParametersPtrOnGPU.resize(2);

			//Affine�p�����[�^
			DataArray& convParam = mParametersPtrOnGPU[0];
			DataArray& convDParam = mDParametersPtrOnGPU[0];

			convParam.size = convDParam.size = mFilterNum * mIcFhFw;

			MALLOC_ON_GPU(convParam);
			MALLOC_ON_GPU(convDParam);

			INITIALIZE_GPU_DATA_NORMAL(convParam, 1, mConvolutionParamWeight);
			INITIALIZE_GPU_DATA_0(convDParam);


			//Bias�p�����[�^
			DataArray& biasParam = mParametersPtrOnGPU[1];
			DataArray& biasDParam = mDParametersPtrOnGPU[1];

			biasParam.size = biasDParam.size = mOutputDataShape.channel;

			MALLOC_ON_GPU(biasParam);
			MALLOC_ON_GPU(biasDParam);

			INITIALIZE_GPU_DATA_0(biasParam);
			INITIALIZE_GPU_DATA_0(biasDParam);


			//�v�Z���ʂ��i�[���邽�߂̃������m��
			mForwardResultOnGPU.setSizeAs4D(mBatchSize, mOc, mOh, mOw);
			mReshapedInputDataOnGPU.setSizeAs3D(mBatchSize, mOhOw, mIcFhFw);
			mBackwardResultOnGPU.setSizeAs4D(mBatchSize, mIc, mIh, mIw);



			MALLOC_ON_GPU(mForwardResultOnGPU);
			MALLOC_ON_GPU(mReshapedInputDataOnGPU);
			MALLOC_ON_GPU(mBackwardResultOnGPU);

			INITIALIZE_GPU_DATA_0(mForwardResultOnGPU);
			INITIALIZE_GPU_DATA_0(mReshapedInputDataOnGPU);
			INITIALIZE_GPU_DATA_0(mBackwardResultOnGPU);
		}

		void Convolution::forwardOnGPU()
		{
			////�����I��groupShared�𗘗p�����ق��������Ɗm�肵���炱�̕����̌�҂Œu��������\��B
			////			{
			////				std::chrono::system_clock::time_point time = std::chrono::system_clock::now();
			////#if 0
			////				dim3 block(16, 16);
			////				dim3 grid(
			////					(mOutputSize + block.x - 1) / block.x,
			////					(mBatchSize + block.y - 1) / block.y);
			////
			////
			////				AffineForward << <grid, block >> > (
			////					mForwardResultOnGPU.address,
			////					mParametersPtrOnGPU[0].address,
			////					mInputDataOnGPU->address,
			////					mParametersPtrOnGPU[1].address,
			////					mOutputSize,
			////					mInputSize,
			////					mBatchSize);
			////
			////#if _DEBUG
			////				CHECK(hipDeviceSynchronize());
			////#endif
			////
			////#else
			////				u32 sharedMemorySize = 48000 / sizeof(f32);
			////
			////
			////				const u32 BlockSize = std::min(static_cast<u32>(1 << 5), sharedMemorySize / (2 * mInputSize));
			////				if (BlockSize < 1)
			////				{
			////					std::cout << "BlockSize is less than 1\n";
			////					assert(0);
			////				}
			////				dim3 block(BlockSize, BlockSize);
			////				dim3 grid((mOutputSize + BlockSize - 1) / BlockSize, (mBatchSize + BlockSize - 1) / BlockSize);
			////
			////				AffineForwardWithSM << <grid, block, 2 * mInputSize * BlockSize * sizeof(f32) >> > (
			////					mForwardResultOnGPU.address,
			////					mParametersPtrOnGPU[0].address,
			////					mInputDataOnGPU->address,
			////					mParametersPtrOnGPU[1].address,
			////					mOutputSize,
			////					mInputSize,
			////					mBatchSize);
			////
			////#if _DEBUG
			////				CHECK(hipDeviceSynchronize());
			////#endif
			////
			////#endif
			////				auto time2 = static_cast<f32>(std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - time).count() / 1000.0f);
			////				//std::cout << time2 << std::endl;
			////				return;
			////		}


			//std::chrono::system_clock::time_point time;


			//if (mWhich == 0)
			//{
			//	if (mNowComparing)
			//	{
			//		time = std::chrono::system_clock::now();
			//	}
			//	dim3 block(32, 32);
			//	dim3 grid(
			//		(mOutputSize + block.x - 1) / block.x,
			//		(mBatchSize + block.y - 1) / block.y);


			//	AffineForward << <grid, block >> > (
			//		mForwardResultOnGPU.address,
			//		mParametersPtrOnGPU[0].address,
			//		mInputDataOnGPU->address,
			//		mParametersPtrOnGPU[1].address,
			//		mOutputSize,
			//		mInputSize,
			//		mBatchSize);

			//	if (mNowComparing)
			//	{
			//		CHECK(hipDeviceSynchronize());
			//		auto elapsedTime = static_cast<f32>(std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - time).count() / 1000.0f);

			//		mFunc0AveTime = (mFunc0AveTime * mFunc0CallCnt + elapsedTime) / (mFunc0CallCnt + 1);
			//		mFunc0CallCnt++;

			//		if (mFunc0CallCnt >= CaptureTimes)
			//		{
			//			mWhich = 1;
			//		}
			//	}
			//}
			//else if (mWhich = 1)
			//{
			//	if (mNowComparing)
			//	{
			//		time = std::chrono::system_clock::now();
			//	}

			//	u32 sharedMemorySize = 48000 / sizeof(f32);


			//	const u32 BlockSize = std::min(static_cast<u32>(1 << 5), sharedMemorySize / (2 * mInputSize));
			//	if (BlockSize < 1)
			//	{
			//		std::cout << "BlockSize is less than 1\n";
			//		assert(0);
			//	}
			//	dim3 block(BlockSize, BlockSize);
			//	dim3 grid((mOutputSize + BlockSize - 1) / BlockSize, (mBatchSize + BlockSize - 1) / BlockSize);

			//	AffineForwardWithSM << <grid, block, 2 * mInputSize * BlockSize * sizeof(f32) >> > (
			//		mForwardResultOnGPU.address,
			//		mParametersPtrOnGPU[0].address,
			//		mInputDataOnGPU->address,
			//		mParametersPtrOnGPU[1].address,
			//		mOutputSize,
			//		mInputSize,
			//		mBatchSize);

			//	if (mNowComparing)
			//	{
			//		CHECK(hipDeviceSynchronize());
			//		auto elapsedTime = static_cast<f32>(std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - time).count() / 1000.0f);

			//		mFunc1AveTime = (mFunc1AveTime * mFunc1CallCnt + elapsedTime) / (mFunc1CallCnt + 1);
			//		mFunc1CallCnt++;

			//		if (mFunc1CallCnt >= CaptureTimes)
			//		{
			//			if (mFunc1AveTime > mFunc0AveTime)
			//			{
			//				mWhich = 0;
			//			}
			//			else
			//			{
			//				mWhich = 1;
			//			}
			//			mNowComparing = false;
			//		}
			//	}

			//}
			//else
			//{
			//	assert(0);
			//}
		}

		void Convolution::backwardOnGPU()
		{
//			//dout�̋t�`��
//			{
//				dim3 block(16, 16);
//				dim3 grid(
//					(mInputSize + block.x - 1) / block.x,
//					(mBatchSize + block.y - 1) / block.y);
//				doutBackward << <grid, block >> > (
//					mBackwardResultOnGPU.address,
//					mParametersPtrOnGPU[0].address,
//					mDInputDataOnGPU->address,
//					mOutputSize,
//					mInputSize,
//					mBatchSize);
//#if _DEBUG
//				CHECK(hipDeviceSynchronize());
//#endif
//			}
//
//			//A�̋t�`��
//			{
//				dim3 block(16, 16);
//				dim3 grid(
//					(mInputSize + block.x - 1) / block.x,
//					(mOutputSize + block.y - 1) / block.y);
//
//				AffineBackward << <grid, block >> > (
//					pDParametersOnGPU[0].address,
//					mDInputDataOnGPU->address,
//					mInputDataOnGPU->address,
//					mOutputSize,
//					mInputSize,
//					mBatchSize);
//
//#if _DEBUG
//				CHECK(hipDeviceSynchronize());
//#endif
//			}
//
//			//Bias�̋t�`��
//			{
//				dim3 block(16);
//				dim3 grid((mOutputSize + block.x - 1) / block.x);
//
//				biasBackward << <grid, block >> > (
//					pDParametersOnGPU[1].address,
//					mDInputDataOnGPU->address,
//					mOutputSize,
//					mBatchSize);
//
//#if _DEBUG
//				CHECK(hipDeviceSynchronize());
//#endif
//			}
		}

		void Convolution::terminateOnGPU()
		{

		}

	}
}