#include <iostream>
#include <thread>
#include <random>
#include <cassert>

#include "./Layer/Layer.h"
#include "./Optimizer/Optimizer.h"
#include "AI.h"
#include "../commonGPU.cuh"
#include "../commonCPU.h"



namespace Aoba
{
#pragma region public

	AI::AI() = default;
	AI::~AI() = default;


	void AI::build(DataFormat4DeepLearning& format, std::unique_ptr<optimizer::BaseOptimizer>&& optimizer, std::unique_ptr<lossFunction::BaseLossFunction>&& lossFunction)
	{
		//�w���Œ�ł�����邩�̃`�F�b�N
		assert(mLayerList.size() > 0);

		//�I�v�e�B�}�C�U�[�̓o�^
		assert(optimizer != nullptr);
		mOptimizer = std::move(optimizer);

		//�����֐��̓o�^
		assert(lossFunction != nullptr);
		mLossFunction = std::move(lossFunction);

		//GPU�̗��p���\���`�F�b�N���A�����o�́B�܂��S�Ă̑w�ɂ��̏��𑗂�B
		checkGpuIsAvailable();

		//�w�̃��������\�������ŕK�v�ɂȂ�p�����[�^�̐ݒ���s���B
		mDataFormat4DeepLearning = format;
		setupLayerInfo(mDataFormat4DeepLearning.trainingDataShape);

		//�e�w���ɂ����郁�����̊m��
		allocLayerMemory();

	}


	void AI::deepLearning(f32* pTrainingData, f32* pTrainingLabel, u32 epochs, f32 learningRate)
	{
		mOptimizer->setLearningRate(learningRate);

		dataSetup(pTrainingData, pTrainingLabel);

		//
		//���`���p�̃f�[�^�������ŏ�������B
		//
		auto printer = [](std::string name, u32 value, u32 stringLen = 15)
		{
			u32 res = stringLen - name.length();
			std::string space = std::string(res, ' ');
			std::cout << name << space << " = " << value << "\n";
		};
		std::cout << "TrainingData setup now" << std::endl;
		printer("TotalData num", mDataFormat4DeepLearning.dataNum);
		printer("channel", mDataFormat4DeepLearning.trainingDataShape.channel);
		printer("height", mDataFormat4DeepLearning.trainingDataShape.height);
		printer("width", mDataFormat4DeepLearning.trainingDataShape.width);

		u32 loopTime = mDataFormat4DeepLearning.dataNum / mDataFormat4DeepLearning.trainingDataShape.batchSize;
		u32 batch = mDataFormat4DeepLearning.trainingDataShape.batchSize;
		auto progressBar = [](u32 currentLoop, u32 totalLoop, u32 length = 20)
		{
			u32 grid = totalLoop / length;
			std::string s = "\r";
			for (u32 i = 0; i < static_cast<u32>((static_cast<f32>(length) * currentLoop) / totalLoop); i++)
			{
				s += "=";
			}
			s += ">";
			int spaceLength = static_cast<s32>(length - s.length() + 1);
			for (s32 i = 0; i < spaceLength; i++)
			{
				s += " ";
			}
			s += " " + std::to_string(static_cast<u32>(static_cast<f32>(currentLoop * 100) / totalLoop)) + "/100";
			printf(s.c_str());
		};

		for (u32 epoch = 0; epoch < epochs; epoch++)
		{
			std::cout << "epoch = " << epoch + 1 << std::endl;
			f32 loss = 0.0f;
			std::cout << "deep learning now" << std::endl;
			for (u32 loop = 0; loop < loopTime; loop++)
			{
				progressBar(loop+1, loopTime);
				u32 offsetForTrainingData = (batch * mDataFormat4DeepLearning.eachTrainingDataSize) * loop;
				u32 offsetForCorrectLabel = (batch * mDataFormat4DeepLearning.eachCorrectDataSize) * loop;
				if (mIsGpuAvailable)
				{
					mInputTrainingDataOnGPU.address = mInputTrainingDataStartAddressOnGPU + offsetForTrainingData;
					mInputLabelDataOnGPU.address = mInputTrainingLableStartAddressOnGPU + offsetForCorrectLabel;
				}
				else
				{
					mInputTrainingDataOnCPU.address = mInputTrainingDataStartAddressOnCPU + offsetForTrainingData;
					mInputLabelDataOnCPU.address = mInputTrainingLableStartAddressOnCPU + offsetForCorrectLabel;
				}
				forward();
#if _DEBUG
				//
				//�����Ő������`�F�b�N
				//
#endif



				backward();
#if _DEBUG
				//
				//�����Ő������`�F�b�N
				//
#endif


				optimize();
#if _DEBUG
				//
				//�����Ő������`�F�b�N
				//
#endif
				if (mIsGpuAvailable)
					loss += mLossOnGPU;
				else
					loss += mLossOnCPU;
			}
			std::cout << "\n";
			std::cout << "current loss = " << loss / loopTime << "\n" << std::endl;
		}
	}


	DataMemory AI::operator()(f32* inputData)
	{
		return DataMemory();
	}

#pragma endregion

#pragma region private
	void AI::checkGpuIsAvailable()
	{
		mIsGpuAvailable = true;
	}

	/// <summary>
	/// �e�w�̓����p�����[�^���v�Z����B
	/// flowDataShape�ɂ͓��̓f�[�^�̌`�󂪓����Ă���̂ŁA
	/// �������ɃJ�[�l���̃T�C�Y��p�����[�^�̐����v�Z�B
	/// </summary>
	void AI::setupLayerInfo(InputDataShape& shape)
	{
		InputDataShape dataShape;
		{
			dataShape.batchSize = shape.batchSize;
			dataShape.channel = shape.channel;
			dataShape.height = shape.height;
			dataShape.width = shape.width;
		}


		for (auto& layer : mLayerList)
		{
			layer->setupLayerInfo(&dataShape);
		}

		mLossFunction->setupDataShape(dataShape);

		if (mIsGpuAvailable)
		{
			mInputTrainingDataOnGPU.size = shape.batchSize * shape.channel * shape.height * shape.width;
		}
		else
		{
			mInputTrainingDataOnCPU.size = shape.batchSize * shape.channel * shape.height * shape.width;
		}
	}

	/// <summary>
	///�@�e�w�ɂ�����p�����[�^�̂��߂̃������m�ۂ⏉�����A
	/// �����Ċw�K���Ɋe�w���K�v�ƂȂ�O�̑w�̏o�̓f�[�^�̃A�h���X��o�^�B
	/// </summary>
	void AI::allocLayerMemory()
	{
		if (mIsGpuAvailable)
		{
			//GPU��̃������̊m�ۂ₻��̏�����
			for (auto& layer : mLayerList)
			{
				layer->initializeOnGPU();
			}

			mLossFunction->initializeOnGPU();

			//�w�K���̊e�w���Q�Ƃ���O�w�̃f�[�^�̃A�h���X��o�^
			//�܂���_�ƂȂ�f�[�^���Z�b�g
			DataMemory* pInputDataOnGPU = &mInputTrainingDataOnGPU;

			//�����Ŋe�w�ɎQ�Ƃ���ׂ��f�[�^�����ɓn���Ă����B
			for (auto& layer : mLayerList)
			{
				layer->setInputDataOnGPU(pInputDataOnGPU);
			}

			//�����֐��ɓn������A���̑��p�r�̂��߂ɏ��`���̏o�͂������ɃZ�b�g����B
			mForwardResultOnGPU = pInputDataOnGPU;

			//�����֐��ɏ��`���̌��ʂ�n���B
			mLossFunction->setInputOnGPU(mForwardResultOnGPU, &mInputLabelDataOnGPU);


			DataMemory* pDInputDataOnGPU = mLossFunction->getDInputDataOnGPU();

			for (auto rit = mLayerList.rbegin(); rit != mLayerList.rend(); rit++)
			{
				(*rit)->setDInputDataOnGPU(pDInputDataOnGPU);
			}
		}
		else
		{
			//GPU��̃������̊m�ۂ₻��̏�����
			for (auto& layer : mLayerList)
			{
				layer->initializeOnCPU();
			}

			mLossFunction->initializeOnCPU();

			//�w�K���̊e�w���Q�Ƃ���O�w�̃f�[�^�̃A�h���X��o�^
			//�܂���_�ƂȂ�f�[�^���Z�b�g
			DataMemory* pInputDataOnCPU = &mInputTrainingDataOnCPU;

			//�����Ŋe�w�ɎQ�Ƃ���ׂ��f�[�^�����ɓn���Ă����B
			for (auto& layer : mLayerList)
			{
				layer->setInputDataOnCPU(pInputDataOnCPU);
			}

			//�����֐��ɓn������A���̑��p�r�̂��߂ɏ��`���̏o�͂������ɃZ�b�g����B
			mForwardResultOnCPU = pInputDataOnCPU;

			//�����֐��ɏ��`���̌��ʂ�n���B
			mLossFunction->setInputOnCPU(mForwardResultOnCPU, &mInputLabelDataOnCPU);


			DataMemory* pDInputDataOnCPU = mLossFunction->getDInputDataOnCPU();

			for (auto rit = mLayerList.rbegin(); rit != mLayerList.rend(); rit++)
			{
				(*rit)->setDInputDataOnCPU(pDInputDataOnCPU);
			}
		}
	}

	void AI::dataSetup(f32* pTrainingData, f32* pTrainingLabel)
	{
		mInputTrainingDataStartAddressOnCPU = pTrainingData;
		mInputTrainingLableStartAddressOnCPU = pTrainingLabel;

		u32 batch = mDataFormat4DeepLearning.trainingDataShape.batchSize;
		if (mIsGpuAvailable)
		{
			mInputTrainingDataOnGPU.size = batch * mDataFormat4DeepLearning.eachTrainingDataSize;
			mInputTrainingDataOnGPU.byteSize = mInputTrainingDataOnGPU.size * sizeof(f32);
			mInputLabelDataOnGPU.size = batch * mDataFormat4DeepLearning.eachCorrectDataSize;
			mInputLabelDataOnGPU.byteSize = mInputLabelDataOnGPU.size * sizeof(f32);

			CHECK(hipMalloc((void**)(&mInputTrainingDataStartAddressOnGPU), mDataFormat4DeepLearning.dataNum * mDataFormat4DeepLearning.eachTrainingDataSize * sizeof(f32)));
			CHECK(hipMalloc((void**)(&mInputTrainingLableStartAddressOnGPU), mDataFormat4DeepLearning.dataNum * mDataFormat4DeepLearning.eachCorrectDataSize * sizeof(f32)));

			CHECK(hipMemcpy(mInputTrainingDataStartAddressOnGPU, mInputTrainingDataStartAddressOnCPU, mDataFormat4DeepLearning.dataNum * mDataFormat4DeepLearning.eachTrainingDataSize * sizeof(f32), hipMemcpyHostToDevice));
			CHECK(hipMemcpy(mInputTrainingLableStartAddressOnGPU, mInputTrainingLableStartAddressOnCPU, mDataFormat4DeepLearning.dataNum * mDataFormat4DeepLearning.eachCorrectDataSize * sizeof(f32), hipMemcpyHostToDevice));
		}
		else
		{
			mInputTrainingDataOnCPU.size = batch * mDataFormat4DeepLearning.eachTrainingDataSize;
			mInputTrainingDataOnCPU.byteSize = mInputTrainingDataOnCPU.size * sizeof(f32);
			mInputLabelDataOnCPU.size = batch * mDataFormat4DeepLearning.eachCorrectDataSize;
			mInputLabelDataOnCPU.byteSize = mInputLabelDataOnCPU.size * sizeof(f32);
		}
	}

	void AI::forward()
	{
		if (mIsGpuAvailable)
		{
			//���`��
			for (auto& layer : mLayerList)
			{
				layer->forwardOnGPU();
			}

			//�����̌v�Z
			mLossOnGPU = mLossFunction->calcLossAndDInputOnGPU();
		}
		else
		{
			//���`��
			for (auto& layer : mLayerList)
			{
				layer->forwardOnCPU();
			}

			//�����̌v�Z
			mLossOnCPU = mLossFunction->calcLossAndDInputOnCPU();
		}
	}

	void AI::backward()
	{
		if (mIsGpuAvailable)
		{
			for (auto riter = mLayerList.rbegin(), end = mLayerList.rend(); riter != end; riter++)
			{
				(*riter)->backwardOnGPU();
			}
		}
		else
		{
			for (auto riter = mLayerList.rbegin(), end = mLayerList.rend(); riter != end; riter++)
			{
				(*riter)->backwardOnCPU();
			}
		}
	}


	void AI::optimize()
	{
		if (mIsGpuAvailable)
		{
			for (auto& layer : mLayerList)
			{
				mOptimizer->optimizeOnGPU(layer);
			}
		}
		else
		{
			for (auto& layer : mLayerList)
			{
				mOptimizer->optimizeOnCPU(layer);
			}
		}

	}

#pragma endregion
}