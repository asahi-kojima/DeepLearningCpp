#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "Sgd.h"




namespace Aoba
{
	namespace
	{
		__global__ void OptimizeOnGPU(f32* param, f32* dParam, f32 learningRate, u32 size)
		{
			u32 id = blockIdx.x * blockDim.x + threadIdx.x;
			if (id >= size)
			{
				return;
			}
			param[id] -= dParam[id] * learningRate;
		}
	}
	namespace optimizer
	{
		void Sgd::optimizeOnGPU(std::unique_ptr<layer::BaseLayer>& pLayer)
		{
			std::vector<DataArray>& params = getLayerParamOnGPU(pLayer);
			std::vector<DataArray>& dParams = getLayerDParamOnGPU(pLayer);

			for (u32 idx = 0; idx < params.size(); idx++)
			{
				DataArray& param = params[idx];
				DataArray& dParam = dParams[idx];

				dim3 block(32);
				dim3 grid((param.size + block.x - 1) / block.x);

				OptimizeOnGPU << <grid, block >> > (param.address, dParam.address, mLearningRate, param.size);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}
		}
	}
}