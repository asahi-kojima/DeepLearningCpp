#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Adam.h"
#include "../../AIHelperFunction.h"

namespace Aoba
{
	namespace
	{
		__global__ void OptimizeOnGPU(f32* param, f32* dParam, f32* m, f32* v, f32 beta0, f32 beta1, f32 learningRate, u32 size)
		{
			u32 id = blockIdx.x * blockDim.x + threadIdx.x;
			if (id >= size)
			{
				return;
			}

			auto dP = dParam[id];
			f32 tmpM = m[id] += (1 - beta0) * (dP - m[id]);
			f32 tmpV = v[id] += (1 - beta1) * (dP * dP - v[id]);

			param[id] -= learningRate * tmpM / (std::sqrtf(tmpV) + 1e-7);
		}
	}


	namespace optimizer
	{
		void Adam::initializeOnGPU(std::vector<std::unique_ptr<layer::BaseLayer> >& layerList)
		{
			const u32 layerSize = layerList.size();

			mMomentumOnGPU.resize(layerSize);
			mVelocityOnGPU.resize(layerSize);

			for (u32 order = 0; order < layerSize; order++)
			{
				auto pLayer = layerList[order].get();
				mLayerOrderMapOnGPU[pLayer] = order;

				std::vector<DataArray>& parameters = getLayerParamOnGPU(layerList[order]);
				mMomentumOnGPU[order].resize(parameters.size());
				mVelocityOnGPU[order].resize(parameters.size());

				for (u32 id = 0; id < mMomentumOnGPU[order].size(); id++)
				{
					mMomentumOnGPU[order][id].size = parameters[id].size;
					mVelocityOnGPU[order][id].size = parameters[id].size;
					MALLOC_AND_INITIALIZE_0_ON_GPU(mMomentumOnGPU[order][id]);
					MALLOC_AND_INITIALIZE_0_ON_GPU(mVelocityOnGPU[order][id]);
				}
			}
		}

		void Adam::optimizeOnGPU(std::unique_ptr<layer::BaseLayer>& refLayer)
		{
			auto pLayer = refLayer.get();
			const u32 order = mLayerOrderMapOnGPU[pLayer];


			std::vector<DataArray>& momentumLst = mMomentumOnGPU[order];
			std::vector<DataArray>& velocityLst = mVelocityOnGPU[order];

			mIteration++;
			f32 effectiveLr = mLearningRate * std::sqrtf(1.0f - std::powf(mBeta1, mIteration)) / (1.0f - std::powf(mBeta0, mIteration));
			for (u32 id = 0; id < momentumLst.size(); id++)
			{
				auto& m = momentumLst[id];
				auto& v = velocityLst[id];

				auto& param = getLayerParamOnGPU(refLayer)[id];
				auto& dParam = getLayerDParamOnGPU(refLayer)[id];

				dim3 block(32);
				dim3 grid((param.size + block.x - 1) / block.x);
				OptimizeOnGPU << <grid, block >> > (param.address, dParam.address, m.address, v.address, mBeta0, mBeta1, effectiveLr, param.size);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}
		}
	}
}