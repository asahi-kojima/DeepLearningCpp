#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>

#include "Affine.h"
#include "../../commonGPU.cuh"

namespace miduho {
	namespace layer
	{
		void Affine::forwardOnGPU(flowDataType** ppFlowData)
		{

		}

		void Affine::backwardOnGPU()
		{

		}

		void Affine::setupParamOnGPU()
		{
			pParametersOnGPU.resize(2);
			pDParametersOnGPU.resize(2);

			//Affine�p�����[�^
			paramMemory& affineParam = pParametersOnGPU[0];
			paramMemory& affineDParam = pDParametersOnGPU[0];

			affineParam.paramNum = affineDParam.paramNum = mOutputSize * mInputSize;

			CHECK(hipMalloc((void**)(&(affineParam.paramAddress)), affineParam.paramNum * sizeof(parameterType))   );
			CHECK(hipMalloc((void**)(&(affineDParam.paramAddress)), affineDParam.paramNum * sizeof(parameterType)) );

			parameterType* tmpAffineParam = new parameterType[affineParam.paramNum];
			{
				std::random_device seed_gen;
				std::default_random_engine engine(seed_gen());
				std::normal_distribution<> dist(0.0, std::sqrt(2.0 / mInputSize));

				parameterType* tmp = new parameterType[affineParam.paramNum];
				for (u32 idx = 0; idx < affineParam.paramNum; idx++)
				{
					tmp[idx] = mAffineParamWeight * static_cast<f32>(dist(engine)) / std::sqrt(2.0f / mInputSize);
				}
				CHECK(hipMemcpy(affineParam.paramAddress, tmp, affineParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));

				for (u32 idx = 0; idx < affineDParam.paramNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(affineDParam.paramAddress, tmp, affineDParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));
				delete[] tmp;
			}


			//Bias�p�����[�^
			paramMemory& biasParam = pParametersOnGPU[1];
			paramMemory& biasDParam = pDParametersOnGPU[1];

			biasParam.paramNum = biasDParam.paramNum = mOutputSize;

			hipMalloc((void**)(&(biasParam.paramAddress)), biasParam.paramNum * sizeof(parameterType));
			hipMalloc((void**)(&(biasDParam.paramAddress)), biasDParam.paramNum * sizeof(parameterType));
			{
				parameterType* tmp = new parameterType[biasParam.paramNum];
				for (u32 idx = 0; idx < biasParam.paramNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(biasParam.paramAddress, tmp, biasParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(biasDParam.paramAddress, tmp, biasDParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			//�v�Z���ʂ��i�[���邽�߂̃������m��
			mForwardResultOnGPU.dataNum = mBatchSize * mOutputSize;
			mBackwardResultOnGPU.dataNum = mBatchSize * mInputSize;
			hipMalloc((void**)(&(mForwardResultOnGPU.dataAddress)), 
				mForwardResultOnGPU.dataNum * sizeof(flowDataType));
			hipMalloc((void**)(&(mBackwardResultOnGPU.dataAddress)), 
				mBackwardResultOnGPU.dataNum * sizeof(flowDataType));
			{
				flowDataType* tmp = new flowDataType[mForwardResultOnGPU.dataNum];
				for (u32 idx = 0; idx < mForwardResultOnGPU.dataNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mForwardResultOnGPU.dataAddress, tmp, 
					mForwardResultOnGPU.dataNum * sizeof(flowDataType), hipMemcpyHostToDevice));
				delete[] tmp;


				tmp = new flowDataType[mBackwardResultOnGPU.dataNum];
				for (u32 idx = 0; idx < mBackwardResultOnGPU.dataNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mBackwardResultOnGPU.dataAddress, tmp, 
					mBackwardResultOnGPU.dataNum * sizeof(flowDataType), hipMemcpyHostToDevice));
				delete[] tmp;
			}
		}
	}
}