#include "hip/hip_runtime.h"
#include <random>
#include <hip/hip_runtime.h>

#include "Affine.h"
#include "../../commonGPU.cuh"
#include <cassert>

namespace miduho {
	namespace layer
	{
		namespace
		{
			using flowDataType = BaseLayer::flowDataType;


			__global__ void AffineForward(
				flowDataType* y, flowDataType* A, 
				flowDataType* x, flowDataType* b, u32 outputSize, u32 inputSize, u32 batchSize)
			{
				u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
				u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
				if (xid >= outputSize || yid >= batchSize)
				{
					return;
				}
				u32 id = yid * outputSize + xid;

				f32 result = 0.0f;
				for (u32 i = 0; i < inputSize; i++)
				{
#if _DEBUG
					u32 tmp = xid * inputSize + i;
					if (tmp < 0 || tmp >= inputSize * outputSize)
					{
						printf("Affine A parameter : out of range : %d\n", tmp);
						printf("threadId x = %d  ,  y = %d\n", threadIdx.x, threadIdx.y);
						assert(0);
					}
					tmp = yid * inputSize + i;
					if (tmp < 0 || tmp >= inputSize * batchSize)
					{
						printf("Affine x parameter : out of range : %d", tmp);
						assert(0);
					}
#endif
					result += A[xid * inputSize + i] * x[yid * inputSize + i]; 
				}
#if _DEBUG
				if (!(id >= 0 && id < batchSize * outputSize))
				{
					printf("Affine y parameter : out of range : %d", id);
					assert(0);
				}
#endif
				y[id] = result + b[xid];
			}

			__global__ void AffineBackward()
			{

			}

			__global__ void biasBackward(flowDataType * dBias, flowDataType * dout, u32 outputSize,u32 batchSize)
			{
				u32 id = blockIdx.x * blockDim.x + threadIdx.x;
				if (id >= outputSize)
				{
					return;
				}
				f32 result = 0.0f;
				for (u32 N = 0; N < batchSize; N++)
				{
#if _DEBUG
					if ((N * outputSize + id) >= batchSize * outputSize)
					{
						assert(0);
					}
#endif
					result += dout[N * outputSize + id];
				}
#if _DEBUG
				if (id >= outputSize)
				{
					assert(0);
				}
#endif
				dBias[id] = result;
			}
		}

		void Affine::forwardOnGPU()
		{
			dim3 block(16,16);
			dim3 grid(
				(mOutputSize + block.x - 1) / block.x,
				(mBatchSize + block.y - 1) / block.y);

			AffineForward << <grid, block >> > (
				mForwardResultOnGPU.dataAddress,
				pParametersOnGPU[0].paramAddress,
				mInputDataOnGPU->dataAddress,
				pParametersOnGPU[1].paramAddress,
				mOutputSize,
				mInputSize,
				mBatchSize);
#if _DEBUG
			CHECK(hipDeviceSynchronize());
#endif
		}

		void Affine::backwardOnGPU()
		{
			{
				dim3 block(16);
				dim3 grid((mOutputSize + block.x - 1) / block.x);
				biasBackward << <grid, block >> > (
					pDParametersOnGPU[1].paramAddress,
					mDInputDataOnGPU->dataAddress,
					mOutputSize,
					mBatchSize);
#if _DEBUG
				CHECK(hipDeviceSynchronize());
#endif
			}
		}

		void Affine::setupParamOnGPU()
		{
			pParametersOnGPU.resize(2);
			pDParametersOnGPU.resize(2);

			//Affine�p�����[�^
			paramMemory& affineParam = pParametersOnGPU[0];
			paramMemory& affineDParam = pDParametersOnGPU[0];

			affineParam.paramNum = affineDParam.paramNum = mOutputSize * mInputSize;

			CHECK(hipMalloc((void**)(&(affineParam.paramAddress)), affineParam.paramNum * sizeof(parameterType))   );
			CHECK(hipMalloc((void**)(&(affineDParam.paramAddress)), affineDParam.paramNum * sizeof(parameterType)) );

			parameterType* tmpAffineParam = new parameterType[affineParam.paramNum];
			{
				std::random_device seed_gen;
				std::default_random_engine engine(seed_gen());
				std::normal_distribution<> dist(0.0, std::sqrt(2.0 / mInputSize));

				parameterType* tmp = new parameterType[affineParam.paramNum];
				for (u32 idx = 0; idx < affineParam.paramNum; idx++)
				{
					tmp[idx] = mAffineParamWeight * static_cast<f32>(dist(engine)) / std::sqrt(2.0f / mInputSize);
				}
				CHECK(hipMemcpy(affineParam.paramAddress, tmp, affineParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));

				for (u32 idx = 0; idx < affineDParam.paramNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(affineDParam.paramAddress, tmp, affineDParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));
				delete[] tmp;
			}


			//Bias�p�����[�^
			paramMemory& biasParam = pParametersOnGPU[1];
			paramMemory& biasDParam = pDParametersOnGPU[1];

			biasParam.paramNum = biasDParam.paramNum = mOutputSize;

			hipMalloc((void**)(&(biasParam.paramAddress)), biasParam.paramNum * sizeof(parameterType));
			hipMalloc((void**)(&(biasDParam.paramAddress)), biasDParam.paramNum * sizeof(parameterType));
			{
				parameterType* tmp = new parameterType[biasParam.paramNum];
				for (u32 idx = 0; idx < biasParam.paramNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(biasParam.paramAddress, tmp, biasParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(biasDParam.paramAddress, tmp, biasDParam.paramNum * sizeof(parameterType), hipMemcpyHostToDevice));
				delete[] tmp;
			}

			//�v�Z���ʂ��i�[���邽�߂̃������m��
			mForwardResultOnGPU.dataNum = mBatchSize * mOutputSize;
			mBackwardResultOnGPU.dataNum = mBatchSize * mInputSize;
			CHECK(hipMalloc((void**)(&(mForwardResultOnGPU.dataAddress)), 
				mForwardResultOnGPU.dataNum * sizeof(flowDataType)));
			CHECK(hipMalloc((void**)(&(mBackwardResultOnGPU.dataAddress)), 
				mBackwardResultOnGPU.dataNum * sizeof(flowDataType)));
			{
				flowDataType* tmp = new flowDataType[mForwardResultOnGPU.dataNum];
				for (u32 idx = 0; idx < mForwardResultOnGPU.dataNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mForwardResultOnGPU.dataAddress, tmp, 
					mForwardResultOnGPU.dataNum * sizeof(flowDataType), hipMemcpyHostToDevice));
				delete[] tmp;


				tmp = new flowDataType[mBackwardResultOnGPU.dataNum];
				for (u32 idx = 0; idx < mBackwardResultOnGPU.dataNum; idx++)
				{
					tmp[idx] = 0.0f;
				}
				CHECK(hipMemcpy(mBackwardResultOnGPU.dataAddress, tmp, 
					mBackwardResultOnGPU.dataNum * sizeof(flowDataType), hipMemcpyHostToDevice));
				delete[] tmp;
			}
		}
	}
}